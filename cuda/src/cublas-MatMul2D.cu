#include "cublas-MatMul2D.cuh"
#include "cublas_utils.cuh"

int MatMul2D(OUT float* C, IN const float* A, const float* B, int M, int N, int K)
{
    // Initialize CUDA and cuBLAS
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLAS initialization failed" << std::endl;
        return 1;
    }

    // Allocate and copy matrices A and B to the GPU
    float* d_A, * d_B, * d_C;
    cudaStat = hipMalloc((void**)&d_A, M * K * sizeof(float));
    cudaStat = hipMalloc((void**)&d_B, K * N * sizeof(float));
    cudaStat = hipMalloc((void**)&d_C, M * N * sizeof(float));
    cudaStat = hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice);

    // Perform GEMM operation
    float alpha = 1.0f;
    float beta = 0.0f;

    // use HIPBLAS_OP_T due to rowmajor tensor and columnmajor cuda

    CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, &alpha, d_A, K, d_B, N, &beta, d_C, M));

    // Copy the result back to the CPU
    cudaStat = hipMemcpy(C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
#if 0
    // Display the result
    std::cout << "Result Matrix C:" << std::endl;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }
#endif
    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);

    return 0;
}